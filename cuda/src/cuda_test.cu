#include "hip/hip_runtime.h"
#include <stdio.h>
#include <vector>

#include <algorithm>
#include <functional>
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <string>
#include <map>
#include <vector>
#include <math.h>
#include <hip/hip_runtime.h>
#include <float.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/sequence.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/replace.h>
#include <thrust/functional.h>
#include <thrust/sort.h> 
#include <thrust/gather.h>
#include <thrust/execution_policy.h>
#include <iostream> 
#include <thrust/binary_search.h>
#include <thrust/random.h>
#include <hiprand.h>
#include <time.h>
#include <stdlib.h>

#define THREADS_PER_BLOCK 256

__global__
void saxpy(int n, float a, float *x, float *y)
{
  if( blockIdx.x==1 && threadIdx.x ==1){
  printf("hello in kernel! \n");
  }
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}

 __device__ __inline__ float my_exp(float a) {
  float b;
  //  b = exp(a);
  b = 2*a;
  return b;
}


struct GenUnifRands
{
    __device__
    float operator () (int idx)
  {
    thrust::default_random_engine randEng;
    thrust::uniform_real_distribution<float> uniDist;
    randEng.discard(idx);
    return uniDist(randEng);
  }
};


__global__
void segmentation_kernel(float* device_xyz,float* device_rgb,int* device_offset,
			 int* neighbor_id,float* device_pdens,int yz_idx,
			 int z_idx,int* parents,float* distances){
  int block_i = blockIdx.x;
  int block_j = blockIdx.y;
  int block_k = blockIdx.z;

  int blockId = yz_idx * block_i + z_idx* block_j + block_k ;
  int threadId = threadIdx.x;

  if( blockId==0 && threadId==0){
    printf("hello in kernel! \n");
  }

  int num_threads = THREADS_PER_BLOCK;
  int my_num_pts = device_offset[blockId + 1] - device_offset[blockId];
  float3 nbr_xyz;
  float3 my_xyz;
  int my_idx;
  int nbr_idx;
  // Iterate over all points in voxel, num_threads points at a time.


  for(int i = 0 ; i < my_num_pts ; i+= num_threads){

    if(threadId + i > my_num_pts){
      return;
    }
    my_idx = device_offset[blockId] + i + threadId;
    //my_idx = 0;
    my_xyz = *(float3*) &device_xyz[3*my_idx];
   

    // Iterate over the neighbouring blocks including yourself.
    float min_distance = 10000000.0;
    float current_parent = my_idx;
    
    for(int j = 0 ; j < 7 ; j++){
      if(neighbor_id[7*blockId+j] == -1){
	continue;
      }

      int nbr_num_pts = device_offset[neighbor_id[7*blockId +j] + 1] - device_offset[neighbor_id[7*blockId + j]];
      
      
      // Iterate over all the points in the neighbouring block.
      for(int k = 0; k < nbr_num_pts; k++){

	nbr_idx = device_offset[neighbor_id[blockId*7 + j]] + k;
	//	nbr_idx = 1;
	nbr_xyz = *(float3*) &device_xyz[3*nbr_idx];

	if(nbr_idx == my_idx){
	  continue;
	}

	float xyz_dist =  pow(my_xyz.x - nbr_xyz.x,2.0) + pow((my_xyz.y - nbr_xyz.y),2.0) +
	  pow((my_xyz.z - nbr_xyz.z),2.0);
	
	if(device_pdens[nbr_idx] > device_pdens[my_idx] && xyz_dist < min_distance){
	  min_distance = xyz_dist;
	  current_parent = nbr_idx;
	  //	  printf("I am here!");
	}
	
      }

    }
      
        distances[my_idx] = min_distance;
    parents[my_idx] = current_parent;
  }

}


__global__
void sampling(float *device_xyz, float *device_rgb, int *device_offset, 
              int *neighbor_id, int yz_idx, int z_idx, float *imp_wt, 
              float* pdensity)
{
  int block_i = blockIdx.x;
  int block_j = blockIdx.y;
  int block_k = blockIdx.z;

  int blockId = yz_idx * block_i + z_idx* block_j + block_k ;
  int threadId = threadIdx.x;
  
  if( blockId==0 && threadId==0){
    printf("hello in kernel! \n");
  }

  int num_threads = THREADS_PER_BLOCK;

  float pdensity_sum = 0.0;
  float nbr_feature[6];
  float norm_sum = 0.0;
  int my_num_pts = device_offset[blockId + 1] - device_offset[blockId];
  float3 nbr_rgb;
  float3 nbr_xyz;
  float3 my_rgb;
  float3 my_xyz;
  float sigma_sq = 0.00005;
  
  float Aij_ew;
  float Aij_ew2;
  // Iterate over all points in voxel, num_threads points at a time. 
    int sum_nbr_num_pts = 0;
  for(int i = 0 ; i < my_num_pts ; i+= num_threads){
    pdensity_sum = 0.0;
    norm_sum = 0.0;
    nbr_feature[0] = 0.0;
    nbr_feature[1] = 0.0;
    nbr_feature[2] = 0.0;
    nbr_feature[3] = 0.0;
    nbr_feature[4] = 0.0;
    nbr_feature[5] = 0.0;

    if(threadId + i > my_num_pts){
      return;
    }

    my_xyz = *(float3*) &device_xyz[3*(device_offset[blockId] + i + threadId)];
    my_rgb = *(float3*) &device_rgb[3*(device_offset[blockId] + i + threadId)];
    
    /*if(blockId == 32 && threadId == 0)
    {
        printf("my x %f, my y %f, my z %f\n", my_xyz.x, my_xyz.y, my_xyz.z);
        printf("my r %f, my g %f, my b %f\n", my_rgb.x, my_rgb.y, my_rgb.z);
    }*/

    // Iterate over the neighbouring blocks including yourself. 
    if(threadId == 0){
    sum_nbr_num_pts = 0;
    }
    for(int j = 0 ; j < 7 ; j++){
      if(neighbor_id[7*blockId+j] == -1){
	    continue;
      }

      int nbr_num_pts = device_offset[neighbor_id[7*blockId +j] + 1] - device_offset[neighbor_id[7*blockId + j]];
      
      if(threadId ==0){
      sum_nbr_num_pts += nbr_num_pts;
      }
      // Iterate over all the points in the neighbouring block.
      for(int k = 0; k < nbr_num_pts; k++){
        
	    nbr_xyz = *(float3*) &device_xyz[3*(device_offset[neighbor_id[blockId*7 + j]] + k)];
	    nbr_rgb = *(float3*) &device_rgb[3*(device_offset[neighbor_id[blockId*7 + j]] + k)];
    
        /*if(blockId == 32 && threadId == 0)
        {
        
  printf("neighbour block = %d, offset = %d\n", neighbor_id[blockId*7 + j],
                                3*(device_offset[neighbor_id[blockId*7+j]] + k)); 
          printf("nbr x %f, nbr y %f, nbr z %f\n", nbr_xyz.x, nbr_xyz.y, nbr_xyz.z);
          printf("nbr r %f, nbr g %f, nbr b %f\n", nbr_rgb.x, nbr_rgb.y, nbr_rgb.z);
        }*/

	    float xyz_dist =  pow(my_xyz.x - nbr_xyz.x,2.0) + pow((my_xyz.y - nbr_xyz.y),2.0) +
                         pow((my_xyz.z - nbr_xyz.z),2.0);
 
        float rgb_dist =  pow((my_rgb.x - nbr_rgb.x),2.0) + pow((my_rgb.y - nbr_rgb.y),2.0) +
						 pow((my_rgb.z - nbr_rgb.z),2.0);
						     
	Aij_ew =  __expf(-1.0f * (xyz_dist/sigma_sq));
	Aij_ew2 = __expf(-1.0f * (xyz_dist + rgb_dist)/sigma_sq); 
	//Aij_ew = pow(xyz_dist,2.0);
	//Aij_ew2 = pow(xyz_dist + rgb_dist,2.0);
	    if(threadId == 1 && blockId==4){
	      printf("xyz_dst: %f , rgb_dst: %f, Aij_ew: %f, Aij_ew2: %f\n",xyz_dist,rgb_dist,Aij_ew, Aij_ew2);
	    }
	    pdensity_sum += Aij_ew2;
		nbr_feature[0] += Aij_ew * nbr_xyz.x;
	    nbr_feature[1] += Aij_ew * nbr_xyz.y;
		nbr_feature[2] += Aij_ew * nbr_xyz.z;
		nbr_feature[3] += Aij_ew * nbr_rgb.x;
	    nbr_feature[4] += Aij_ew * nbr_rgb.y;
	    nbr_feature[5] += Aij_ew * nbr_rgb.z; 
      }
    }
    
    norm_sum += pow(my_xyz.x - nbr_feature[0],2.0);
    norm_sum += pow(my_xyz.y - nbr_feature[1],2.0);
    norm_sum += pow(my_xyz.z - nbr_feature[2],2.0);
    norm_sum += pow(my_rgb.x - nbr_feature[3],2.0);
    norm_sum += pow(my_rgb.y - nbr_feature[4],2.0);
    norm_sum += pow(my_rgb.z - nbr_feature[5],2.0);
    
    pdensity[device_offset[blockId] + i + threadId] = pdensity_sum;

    imp_wt[device_offset[blockId] + i + threadId] = norm_sum;
    
  }
  /*
  if((threadId == 0)){
    printf("my_num_pts: %d ,my_sum_nbr_pts:%d, blockId: %d \n", my_num_pts,sum_nbr_num_pts, blockId );
    }
  */
}


  
  

  


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
  if (code != hipSuccess) 
    {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
    }
}


int device_setup(int num_pts, int num_voxels,   float *flattenXYZ,
		 float *flattenRGB, int *voxel_offset, int *neighbor_ids,
		 int x_idx, int y_idx, int z_idx,int num_samples,uint *sample_arr,float *pdens)
{
  printf("HELLO IN DEVICE SETUP!\n");

  float *device_xyz, *device_rgb;
  int *device_offset, *device_neighbor_ids;

  //  hipMalloc(&device_xyz, num_pts*3*sizeof(float));
  hipMalloc(&device_xyz, num_pts*3*sizeof(float));
  hipMalloc(&device_rgb, num_pts*3*sizeof(float));
  hipMalloc(&device_offset, (num_voxels+1)*sizeof(int));
  hipMalloc(&device_neighbor_ids , num_voxels * 7 * sizeof(int));
  //  hipMalloc(&imp_wt, num_pts*sizeof(float));
  //hipMalloc(&pdensity, num_pts*sizeof(float));
  //host_imp_wt = (float*) malloc(num_pts*sizeof(float));
  //host_pdensity = (float*) malloc(num_pts*sizeof(float));

  thrust::device_vector<float> dev_imp_wt(num_pts);
  thrust::device_vector<float> dev_pdensity(num_pts);
  thrust::host_vector<float> host_imp_wt(num_pts);
  thrust::host_vector<float> host_pdensity(num_pts);
  
  float* imp_wt = thrust::raw_pointer_cast(dev_imp_wt.data());
  float* pdensity = thrust::raw_pointer_cast(dev_pdensity.data());
  printf("finished mallocing!\n");

  


  gpuErrchk(hipMemcpy(device_offset,voxel_offset, (num_voxels+1)*sizeof(int),hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(device_xyz, flattenXYZ, num_pts*3*sizeof(float),hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(device_rgb, flattenRGB, num_pts*3*sizeof(float),hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(device_neighbor_ids, neighbor_ids, num_voxels*7*sizeof(int),hipMemcpyHostToDevice));
  
  int yz_idx = y_idx * z_idx;

  dim3 gridDim(x_idx,y_idx,z_idx);
  dim3 blockDim(THREADS_PER_BLOCK,1,1);
  
  printf("about to call kernel\n");
  sampling<<<gridDim,blockDim>>>(device_xyz,device_rgb,device_offset,device_neighbor_ids,yz_idx,z_idx,imp_wt,pdensity);
  printf("finished sampling!\n");
  
  hipDeviceSynchronize();
  
  thrust::copy(dev_imp_wt.begin(),dev_imp_wt.end(),host_imp_wt.begin());
  thrust::copy(dev_pdensity.begin(),dev_pdensity.end(),host_pdensity.begin());


  //  gpuErrchk(hipMemcpy(host_imp_wt,imp_wt,num_pts*sizeof(float),hipMemcpyDeviceToHost));
  //gpuErrchk(hipMemcpy(host_pdensity,pdensity,num_pts*sizeof(float),hipMemcpyDeviceToHost));
  
  for(int i = 0 ; i < 10; i++){
    printf("imp_wt[%d] = %0.9f pdensity[%d] = %f\n",i,host_imp_wt[i],i,host_pdensity[i]);
  }
  

  /*
  thrust::host_vector<int> h_vec(20); thrust::generate(h_vec.begin(), h_vec.end(), rand);
  for(int i = 0; i < 20 ; i++){
    printf("i:%d, %d\n" , i, h_vec[i]);
  }
  // transfer data to the device
  thrust::device_vector<int> d_vec = h_vec;
  // sort data on the device (805 Mkeys/sec on GeForce GTX 480)
  thrust::sort(d_vec.begin(), d_vec.end()); // transfer data back to host
  thrust::copy(d_vec.begin(), d_vec.end(), h_vec.begin());
  for(int i = 0 ; i < 20 ; i++){
    printf("i:%d, %d\n" , i, h_vec[i]);
}
  */




  // WEIGHTED SAMPLING


  
  printf("====================\n");
  printf("going to do weighted sampling\n");
  // step 1: normalize: (TODO: use functor to make fast )
  float norm_sum = thrust::reduce(dev_imp_wt.begin(),dev_imp_wt.end());
  float norm_factor = 1.0/norm_sum;
  printf("norm_sum: %f\n",norm_sum);

  thrust::device_vector<float> temp(dev_imp_wt.size());
  thrust::fill(temp.begin(),temp.end(),norm_factor);
  thrust::transform(dev_imp_wt.begin(),dev_imp_wt.end(),temp.begin(),dev_imp_wt.begin(),thrust::multiplies<float>());

  thrust::copy(dev_imp_wt.begin(),dev_imp_wt.end(),host_imp_wt.begin());
  printf("\n normalized weights \n");
    for(int i = 0 ; i < 5 ; i++){
      printf("h_wt[%d] = %f , ",i,host_imp_wt[i]);
    }

    // step 2: compute prefix sum (clusive scan)
    thrust::device_vector<float> wts_rs(dev_imp_wt.size());
    thrust::inclusive_scan(dev_imp_wt.begin(),dev_imp_wt.end(),wts_rs.begin());
    printf("\n rolling_sum \n");
    thrust::copy(wts_rs.begin(),wts_rs.end(),host_imp_wt.begin());
    for(int i = 0 ; i < 5 ; i++){
      printf("h_wt[%d] = %0.9f , ",i,host_imp_wt[i]);
    }

    // step 3: generate uniform random numbers:
    srand(time(NULL));
    int seed = rand();
    printf("\n get random samples \n");

    thrust::device_vector<float> d_unifrands(num_samples);
    thrust::transform( thrust::make_counting_iterator(seed), thrust::make_counting_iterator(seed + num_samples),
                       d_unifrands.begin(),GenUnifRands());


    // step 4 : generate (weighted) random samples
    thrust::device_vector<unsigned int> samples(num_samples);
    thrust::lower_bound(wts_rs.begin(),wts_rs.end(),d_unifrands.begin(),d_unifrands.end(),samples.begin());
    thrust::host_vector<unsigned int> h_samples(num_samples);
    thrust::copy(samples.begin(),samples.end(),h_samples.begin());
    for(int i = 0 ; i < 10 ; i++){
      printf("h_samples[%d] = %d " , i , h_samples[i]);
    }
   
    printf("\n");
    uint* pc_samples = thrust::raw_pointer_cast(h_samples.data());
    //uint* samples_arr = (uint*) malloc(num_samples* sizeof(uint));
    memcpy(sample_arr, pc_samples,num_samples*sizeof(uint));
    
    
    thrust::device_vector<float> dev_pdensity_rs(num_samples);
    thrust::host_vector<float> host_pdensity_rs(num_samples);
    thrust::gather(thrust::device,samples.begin(),samples.end(),dev_pdensity.begin(),dev_pdensity_rs.begin());
    thrust::copy(dev_pdensity_rs.begin(),dev_pdensity_rs.end(),host_pdensity_rs.begin());
    float* host_pdensity_rs_ptr = thrust::raw_pointer_cast(host_pdensity_rs.data());
    memcpy(pdens, host_pdensity_rs_ptr,num_samples*sizeof(float));
    
    /*
    for(int i = 0 ; i < 10; i++){
      printf("samples_arr[%d] = %d , " , i ,samples_arr[i] );
    }
    */

    /*
    thrust::host_vector<float> h_unifrands(num_samples);
    thrust::copy(d_unifrands.begin(),d_unifrands.end(),h_unifrands.begin());
    for(int i = 0 ; i < 10 ; i++){
      printf("unifrands[%d] = %f " , i , h_unifrands[i]);
    }
    */

    // step 5: use a gather operator to "compress" xyz rgb into appropriate form:

  return(1);
}






int segmentation(int num_pts, int num_voxels, float* pdens,  float *flattenXYZ,
                 float *flattenRGB, int *voxel_offset, int *neighbor_ids,
                 int x_idx, int y_idx, int z_idx,int* parents_ptr)
{
  printf("HELLO IN SEGMENTATION KERNEL!\n");

  float *device_xyz, *device_rgb, *device_pdens;
  int *device_offset, *device_neighbor_ids;


  hipMalloc(&device_xyz, num_pts*3*sizeof(float));
  hipMalloc(&device_rgb, num_pts*3*sizeof(float));
  hipMalloc(&device_offset, (num_voxels+1)*sizeof(int));
  hipMalloc(&device_neighbor_ids , num_voxels * 7 * sizeof(int));
  hipMalloc(&device_pdens, num_pts*sizeof(float));

  thrust::device_vector<int> dev_parents(num_pts);
  thrust::host_vector<int> host_parents(num_pts);
  thrust::device_vector<float> dev_distances(num_pts);
  thrust::host_vector<float> host_distances(num_pts);

  int* parents = thrust::raw_pointer_cast(dev_parents.data());
  float* distances = thrust::raw_pointer_cast(dev_distances.data());
  printf("finished mallocing!\n");

  gpuErrchk(hipMemcpy(device_offset,voxel_offset, (num_voxels+1)*sizeof(int),hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(device_xyz, flattenXYZ, num_pts*3*sizeof(float),hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(device_rgb, flattenRGB, num_pts*3*sizeof(float),hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(device_neighbor_ids, neighbor_ids, num_voxels*7*sizeof(int),hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(device_pdens, pdens, num_pts * sizeof(float), hipMemcpyHostToDevice));

  int yz_idx = y_idx * z_idx;
  dim3 gridDim(x_idx,y_idx,z_idx);
  dim3 blockDim(THREADS_PER_BLOCK,1,1);

  printf("about to call kernel\n");
  segmentation_kernel<<<gridDim,blockDim>>>(device_xyz,device_rgb,device_offset,device_neighbor_ids,device_pdens,yz_idx,z_idx,parents,distances);
  printf("finished segmentation!\n");
  
  int num_gather = 6;
  thrust::device_vector<int> temp_1(num_pts);
  thrust::device_vector<int> temp_2(num_pts);

  thrust::copy(dev_parents.begin(),dev_parents.end(),temp_1.begin());
  for(int i  = 0 ; i < num_gather ; i++){
    thrust::gather(thrust::device, dev_parents.begin(),dev_parents.end(),temp_1.begin(),temp_2.begin());
    thrust::copy(temp_2.begin(),temp_2.end(),temp_1.begin());
    thrust::copy(temp_2.begin(),temp_2.end(),dev_parents.begin());
  }
  

  printf("finished tree cutting!\n");
  thrust::copy(dev_parents.begin(),dev_parents.end(),host_parents.begin());
   int* host_parents_ptr = thrust::raw_pointer_cast(host_parents.data());
   memcpy(parents_ptr, host_parents_ptr,num_pts*sizeof(int));
   
  for(int i = 0 ; i < 5 ; i++){
    printf("parents[%d] = %d \n", i ,host_parents[i]);
  }
  
  
  printf("DONE!!!!");
  hipDeviceSynchronize();
  return(1);
}
