#include "hip/hip_runtime.h"
#include <stdio.h>
#include <vector>

#include <algorithm>
#include <functional>
#include <math.h>       /* pow */
#include <cstdlib>
#include <string>
#include <map>
#include <vector>

#define THREADS_PER_BLOCK 256

__global__
void saxpy(int n, float a, float *x, float *y)
{
  if( blockIdx.x==1 && threadIdx.x ==1){
  printf("hello in kernel! \n");
  }
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}

__global__
void sampling(float *device_xyz, float *device_rgb, int *device_offset, 
              int *neighbor_id, int xy_idx, int y_idx, float *imp_wt, 
              float* pdensity)
{
  int block_i = blockIdx.x;
  int block_j = blockIdx.y;
  int block_k = blockIdx.z;

  int blockId = xy_idx * block_i + y_idx* block_j + block_k ;
  int threadId = threadIdx.x;
  
  if( blockId==0 && threadId==0){
    printf("hello in kernel! \n");
  }

  int num_threads = THREADS_PER_BLOCK;

  float pdensity_sum = 0.0;
  float nbr_feature[6];
  float norm_sum = 0.0;
  int my_num_pts = device_offset[blockId + 1] - device_offset[blockId];
  float3 nbr_rgb;
  float3 nbr_xyz;
  float3 my_rgb;
  float3 my_xyz;
  float sigma_sq = 0.00005;
  /*if((threadId == 0)){
    printf("my_num_pts: %d , blockId: %d \n", my_num_pts, blockId );
  }*/

  // Iterate over all points in voxel, num_threads points at a time. 
  for(int i = 0 ; i < my_num_pts ; i+= num_threads){
    pdensity_sum = 0.0;
    norm_sum = 0.0;
    nbr_feature[0] = 0.0;
    nbr_feature[1] = 0.0;
    nbr_feature[2] = 0.0;
    nbr_feature[3] = 0.0;
    nbr_feature[4] = 0.0;
    nbr_feature[5] = 0.0;
    
    if(threadId + i > my_num_pts){
      return;
    }

    my_xyz = *(float3*) &device_xyz[3*(device_offset[blockId] + i + threadId)];
    my_rgb = *(float3*) &device_rgb[3*(device_offset[blockId] + i + threadId)];

    /*if(blockId == 32 && threadId == 0)
    {
        printf("my x %f, my y %f, my z %f\n", my_xyz.x, my_xyz.y, my_xyz.z);
        printf("my r %f, my g %f, my b %f\n", my_rgb.x, my_rgb.y, my_rgb.z);
    }*/

    // Iterate over the neighbouring blocks including yourself. 
    for(int j = 0 ; j < 7 ; j++){
      if(neighbor_id[7*blockId+j] == -1){
	    continue;
      }

      int nbr_num_pts = device_offset[neighbor_id[7*blockId +j] + 1] - device_offset[neighbor_id[7*blockId + j]];

      // Iterate over all the points in the neighbouring block.
      for(int k = 0; k < nbr_num_pts; k++){
        
	    nbr_xyz = *(float3*) &device_xyz[3*(device_offset[neighbor_id[blockId*7 + j]] + k)];
	    nbr_rgb = *(float3*) &device_rgb[3*(device_offset[neighbor_id[blockId*7 + j]] + k)];
    
        /*if(blockId == 32 && threadId == 0)
        {
          printf("neighbour block = %d, offset = %d\n", neighbor_id[blockId*7 + j],
                                3*(device_offset[neighbor_id[blockId*7+j]] + k)); 
          printf("nbr x %f, nbr y %f, nbr z %f\n", nbr_xyz.x, nbr_xyz.y, nbr_xyz.z);
          printf("nbr r %f, nbr g %f, nbr b %f\n", nbr_rgb.x, nbr_rgb.y, nbr_rgb.z);
        }*/

	    float xyz_dist = pow(my_xyz.x - nbr_xyz.x,2.0) + pow((my_xyz.y - nbr_xyz.y),2.0) +
                         pow((my_xyz.z - nbr_xyz.z),2.0);
 
        float rgb_dist = pow((my_rgb.x - nbr_rgb.x),2.0) + pow((my_rgb.y - nbr_rgb.y),2.0) +
						 pow((my_rgb.z - nbr_rgb.z),2.0);
						     
        float Aij_ew = exp(-1.0 * xyz_dist/sigma_sq);
	    float Aij_ew2 = exp(-1.0 * (xyz_dist + rgb_dist)/sigma_sq); 
						     
	    pdensity_sum += Aij_ew2;
		nbr_feature[0] += Aij_ew * nbr_xyz.x;
	    nbr_feature[1] += Aij_ew * nbr_xyz.y;
		nbr_feature[2] += Aij_ew * nbr_xyz.z;
		nbr_feature[3] += Aij_ew * nbr_rgb.x;
	    nbr_feature[4] += Aij_ew * nbr_rgb.y;
	    nbr_feature[5] += Aij_ew * nbr_rgb.z; 
      }
    }
    
    norm_sum += pow(my_xyz.x - nbr_feature[0],2.0);
    norm_sum += pow(my_xyz.y - nbr_feature[1],2.0);
    norm_sum += pow(my_xyz.z - nbr_feature[2],2.0);
    norm_sum += pow(my_rgb.x - nbr_feature[3],2.0);
    norm_sum += pow(my_rgb.y - nbr_feature[4],2.0);
    norm_sum += pow(my_rgb.z - nbr_feature[5],2.0);
    
    pdensity[device_offset[blockId] + i + threadId] = pdensity_sum;
    imp_wt[device_offset[blockId] + i + threadId] = norm_sum;
    
  }
}
  
  

  


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
  if (code != hipSuccess) 
    {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
    }
}


int device_setup(int num_pts, int num_voxels,   float *flattenXYZ,
		 float *flattenRGB, int *voxel_offset, int *neighbor_ids,
         int x_idx, int y_idx, int z_idx)
{
  printf("HELLO IN DEVICE SETUP!\n");

  float *device_xyz, *device_rgb, *imp_wt, *pdensity, *host_imp_wt, *host_pdensity;
  int *device_offset, *device_neighbor_ids;

  hipMalloc(&device_xyz, num_pts*3*sizeof(float));
  hipMalloc(&device_xyz, num_pts*3*sizeof(float));
  hipMalloc(&device_rgb, num_pts*3*sizeof(float));
  hipMalloc(&device_offset, (num_voxels+1)*sizeof(int));
  hipMalloc(&device_neighbor_ids , num_voxels * 7 * sizeof(int));
  hipMalloc(&imp_wt, num_pts*sizeof(float));
  hipMalloc(&pdensity, num_pts*sizeof(float));
  host_imp_wt = (float*) malloc(num_pts*sizeof(float));
  host_pdensity = (float*) malloc(num_pts*sizeof(float));

  printf("finished mallocing!\n");

  gpuErrchk(hipMemcpy(device_offset,voxel_offset, (num_voxels+1)*sizeof(int),hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(device_xyz, flattenXYZ, num_pts*3*sizeof(float),hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(device_rgb, flattenRGB, num_pts*3*sizeof(float),hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(device_neighbor_ids, neighbor_ids, num_voxels*7*sizeof(int),hipMemcpyHostToDevice));
  
  int xy_idx = x_idx * y_idx;

  dim3 gridDim(x_idx,y_idx,z_idx);
  dim3 blockDim(THREADS_PER_BLOCK,1,1);
  
  printf("about to call kernel\n");
  sampling<<<gridDim,blockDim>>>(device_xyz,device_rgb,device_offset,device_neighbor_ids,xy_idx,y_idx,imp_wt,pdensity);
  printf("finished sampling!\n");
  
  hipDeviceSynchronize();
  
  gpuErrchk(hipMemcpy(host_imp_wt,imp_wt,num_pts*sizeof(float),hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(host_pdensity,pdensity,num_pts*sizeof(float),hipMemcpyDeviceToHost));

  for(int i = 0 ; i < 100; i++){
    printf("imp_wt[%d] = %f pdensity[%d] = %f\n",i,host_imp_wt[i],i,host_pdensity[i]);
  }

  return(1);
}




