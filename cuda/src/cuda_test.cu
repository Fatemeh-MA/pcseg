#include "hip/hip_runtime.h"
#include <stdio.h>

__global__
void saxpy(int n, float a, float *x, float *y)
{
  if( blockIdx.x==1 && threadIdx.x ==1){
  printf("hello in kernel! \n");
  }
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
  if (code != hipSuccess) 
    {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
    }
}
int run_main(void)
{
  int N = 1<<20;
  float *x, *y, *d_x, *d_y;
  x = (float*)malloc(N*sizeof(float));
  y = (float*)malloc(N*sizeof(float));

  gpuErrchk(hipMalloc(&d_x, N*sizeof(float)));
  gpuErrchk(hipMalloc(&d_y, N*sizeof(float)));

  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  gpuErrchk(hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice));

  // Perform SAXPY on 1M elements
  saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);

  gpuErrchk(hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost));

  float maxError = 0.0f;
  for (int i =0 ; i < 4 ; i++){
    printf("y= %0.4f \n", y[i]);
  }

  for (int i = 0; i < N; i++)
    maxError = max(maxError, abs(y[i]-4.0f));
  printf("Max error: %f\n", maxError);

  hipFree(d_x);
  hipFree(d_y);
  free(x);
  free(y);
  return(1);
}



