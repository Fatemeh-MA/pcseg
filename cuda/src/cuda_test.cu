#include "hip/hip_runtime.h"
#include <stdio.h>
#include <vector>

#include <algorithm>
#include <functional>
#include <math.h>       /* pow */
#include <cstdlib>
#include <string>
#include <map>
#include <vector>
__global__
void saxpy(int n, float a, float *x, float *y)
{
  if( blockIdx.x==1 && threadIdx.x ==1){
  printf("hello in kernel! \n");
  }
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}

__global__
void sampling(float* device_xyz, float* device_rgb, int* device_offset, int* neighbor_id, int xy_idx, int y_idx,float* imp_wt, float* pdensity){
  int block_i = blockIdx.x;
  int block_j = blockIdx.y;
  int block_k = blockIdx.z;

  int blockId = xy_idx * block_i + y_idx* block_j + block_k ;

  if(threadIdx.x ==0 ){
    //    printf("hello!, in block: %d \n", blockId);
  }
  int num_threads = 256;

  float pdensity_sum = 0.0;
  float nbr_feature[6];
  float norm_sum = 0.0;
  int my_num_pts = device_offset[blockId + 1 ] - device_offset[blockId];
  float3 nbr_rgb;
  float3 nbr_xyz;
  float3 my_rgb;
  float3 my_xyz;
  float sigma_sq = 0.00005;
  if(threadIdx.x == 0){
    //    printf("my_num_pts: %d , blockId: %d \n",my_num_pts,blockId );
  }
  for( int i = 0 ; i < my_num_pts ; i+= num_threads){
    pdensity_sum = 0.0;
    norm_sum = 0.0;
    nbr_feature[0] = 0.0;
    nbr_feature[1] = 0.0;
    nbr_feature[2] = 0.0;
    nbr_feature[3] = 0.0;
    nbr_feature[4] = 0.0;
    nbr_feature[5] = 0.0;
    
    if(threadIdx.x + i > my_num_pts){
      break;
    }

    my_xyz = *(float3*) &device_xyz[device_offset[blockId] + i + threadIdx.x];
    my_rgb = *(float3*) &device_rgb[device_offset[blockId] + i + threadIdx.x];  
    
    for(int j = 0 ; j < 7 ; j++){
      if(neighbor_id[7*blockId+j] == -1){
	continue;
      }
      int nbr_num_pts = device_offset[neighbor_id[7*blockId +j] + 1 ] - device_offset[neighbor_id[7*blockId + j] ];

      for(int k = 0 ; k < nbr_num_pts; k++){

	nbr_rgb = *(float3*) &device_rgb[device_offset[neighbor_id[blockId*7 + j]+k]];
	nbr_xyz = *(float3*) &device_xyz[device_offset[neighbor_id[blockId*7 + j]+k]];
	 float xyz_dist = pow(my_xyz.x - nbr_xyz.x,2.0)+ 
						     pow((my_xyz.y - nbr_xyz.y),2.0)
						     +pow((my_xyz.z - nbr_xyz.z),2.0);
						     float rgb_dist = pow((my_rgb.x - nbr_rgb.x),2.0)+
                                                     pow((my_rgb.y - nbr_rgb.y),2.0)
						     +pow((my_rgb.z - nbr_rgb.z),2.0);
						     
						     float Aij_ew = exp(-1.0 * xyz_dist/sigma_sq);
						     float Aij_ew2 = exp(-1.0 * (xyz_dist + rgb_dist)/sigma_sq);						 
						     
						     pdensity_sum += Aij_ew2;
						     nbr_feature[0] += Aij_ew * nbr_xyz.x;
						     nbr_feature[1] += Aij_ew* nbr_xyz.y;
						     nbr_feature[2] += Aij_ew* nbr_xyz.z;
						     nbr_feature[3] += Aij_ew* nbr_rgb.x;
						     nbr_feature[4] += Aij_ew* nbr_rgb.y;
						     nbr_feature[5] += Aij_ew* nbr_rgb.z; 
      }
    }
    
    norm_sum += pow(my_xyz.x - nbr_feature[0],2.0);
    norm_sum += pow(my_xyz.y - nbr_feature[1],2.0);
    norm_sum += pow(my_xyz.z - nbr_feature[2],2.0);
    norm_sum += pow(my_xyz.x - nbr_feature[3],2.0);
    norm_sum += pow(my_xyz.y - nbr_feature[4],2.0);
    norm_sum += pow(my_xyz.z - nbr_feature[5],2.0);
    pdensity[device_offset[blockId] + i + threadIdx.x] = pdensity_sum;
    imp_wt[device_offset[blockId] + i + threadIdx.x ] = norm_sum;
    if(threadIdx.x==0){
      printf("imp_wt: %f\n",norm_sum);
    }
      }
   
    
}
  
  

  


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
  if (code != hipSuccess) 
    {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
    }
}

int device_setup(int num_pts, int num_voxels,   float* flattenXYZ,
		 float* flattenRGB,int* voxel_offset, int* neighbor_ids,int x_idx,int y_idx,int z_idx){


  printf("HELLO IN DEVICE SETUP!\n");

  float *device_xyz, *device_rgb;
  int* device_offset, *device_neighbor_ids;
  float *imp_wt, *pdensity;

    hipMalloc(&device_xyz, num_pts*3*sizeof(float));
  
  hipMalloc(&device_xyz, num_pts*3*sizeof(float));
  hipMalloc(&device_rgb, num_pts*3*sizeof(float));
  hipMalloc(&device_offset, (num_voxels+1)*sizeof(int));
  hipMalloc(&device_neighbor_ids , num_voxels * 7 * sizeof(int));

  hipMalloc(&imp_wt, num_pts*sizeof(float));
  hipMalloc(&pdensity, num_pts*sizeof(float));

  printf("finished mallocing!\n");
  gpuErrchk(hipMemcpy(device_offset,voxel_offset, (num_voxels+1)*sizeof(int),hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(device_xyz, flattenXYZ, num_pts*3*sizeof(float),hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(device_rgb, flattenRGB, num_pts*3*sizeof(float),hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(device_neighbor_ids, neighbor_ids, num_voxels*7*sizeof(int),hipMemcpyHostToDevice));
  int xy_idx = x_idx * y_idx;

  dim3 gridDim(x_idx,y_idx,z_idx);
  dim3 blockDim(256,1,1);
  printf("about to call kernel\n");
  sampling<<<gridDim,blockDim>>>(device_xyz,device_rgb,device_offset,device_neighbor_ids,xy_idx,y_idx,imp_wt,pdensity);
  printf("finished sampling!\n");
  hipDeviceSynchronize();
  float* host_imp_wt;
  host_imp_wt = (float*) malloc(num_pts*sizeof(float));
  gpuErrchk(hipMemcpy(host_imp_wt,imp_wt,num_pts*sizeof(float),hipMemcpyDeviceToHost));

  for(int i = 0 ; i < 100; i++){
    printf("imp_wt[%d] = %f\n",i,host_imp_wt[i]);
  }

  return(1);
}




