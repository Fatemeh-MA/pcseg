#include "hip/hip_runtime.h"
#include <stdio.h>
#include <vector>

#include <algorithm>
#include <functional>
#include <math.h>       /* pow */
#include <cstdlib>
#include <string>
#include <map>
#include <vector>
__global__
void saxpy(int n, float a, float *x, float *y)
{
  if( blockIdx.x==1 && threadIdx.x ==1){
  printf("hello in kernel! \n");
  }
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}


__global__
void sampling(float* device_xyz, float* device_rgb, float* device_offset){
  int block_i = blockIdx.x;
  int block_j = blockIdx.y;
  int block_k = blockIDx.z;

  int blockId = ;
  // do pre-computation of neighbor offsets
  //find all 6 neighbors and offset:
  // be careful for last voxel.
  int neighborhood_size;
  int pts_voxel = device_offset[i+1] - device_offset[i];
  
  // initialize importance weight in shared memory for that voxel.
  // shared memory for importance is of size voxel_stride
  for(int global_iter = 0 ; i < pts_voxel ; i+= voxel_stride)
    
  for(int i = 0 ; i < neighborhood_size; i+=256){
    //load 256 neighbors information into shared memory.    
    for(int j =0 ; j < voxel_stride; j+=256){
      int pt_idx = j + threadIdx.x;
      for (int k = 0 ; k < 256 ; k++){
	float dist = (your_x - nbr_x)^2 +(your_y - nbr_y)^2 + (your_z -nbr_z)^2;
	float ew = exp(-1* dist/sigma_sq);
	float your_feature[0] += ew * your_x
	  float your_feature[1] += ew * your_x
	  float your_feature[2] += ew * your_z
	  ..
      }
    }

    for(int j = 0 ; j < voxelstrid ; j+=256){
    imp_wt[j+threadIdx.x] = norm(x- partial_sum computed)
      }
     
    //finished computing importanve weight for voxel stride
  }


      imp_wt[pt_idx] += ();

    }


  }


//2nd: kernel
//maybe use thrust to find local and global sums

// weighted sampling kernel
void weighted_sampling_kenrel{

  int num_samples = local_sum/global_sum * total_samples;
  bin
  for (int i = 0 ; i < bins ; i+=256){
    //    local sum over 256 strides
    for ( int j = 0 ; j < num_samples* ; j++)
      //each thread will generate a random number
      //      and check if it falls in this particular bin[i]
      // and if it does, then break.
  }

}



  }


  
  

}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
  if (code != hipSuccess) 
    {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
    }
}
int device_setup(int num_pts, int num_voxels,   float* flattenXYZ,
		 float* flattenRGB,int* voxel_offset){


  printf("HELLO IN DEVICE SETUP!\n");

  float *device_xyz, *device_rgb;
  int* device_offset;
  hipMalloc(&device_xyz, num_pts*3*sizeof(float));
  hipMalloc(&device_rgb, num_pts*3*sizeof(float));
  hipMalloc(&device_offset, num_voxels*3*sizeof(int));
  
  gpuErrchk(hipMemcpy(device_offset,voxel_offset, num_voxels*sizeof(int),hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(device_xyz, flattenRGB, num_pts*3*sizeof(float),hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(device_rgb, flattenRGB, num_pts*3*sizeof(float),hipMemcpyHostToDevice));



  float x_grid = 0.5;
  float y_grid = 0.5;
  float z_grid = 0.5;
  float max_xyz = std::max(max_x-min_x,max_y-min_y);
  max_xyz = std::max(max_xyz, max_z-min_z);
  int xy_idx = (max_xyz/x_grid)*(max_xyz/y_grid);
  int x_idx = max_xyz/x_grid;
  int y_idx  = (max_xyz/y_grid);
  int z_idx = (max_xyz/z_grid);

  dim3 gridDim(x_idx,y_idx,z_ids);
  dim3 blockDim(256);

  sampling<<<gridDim,blockDim>>>(device_xyz,device_rgb,device_offset);


}
/*{
  int N = 1<<20;
  float *x, *y, *d_x, *d_y;
  x = (float*)malloc(N*sizeof(float));
  y = (float*)malloc(N*sizeof(float));

  gpuErrchk(hipMalloc(&d_x, N*sizeof(float)));
  gpuErrchk(hipMalloc(&d_y, N*sizeof(float)));

  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  gpuErrchk(hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice));

  // Perform SAXPY on 1M elements
  saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);

  gpuErrchk(hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost));

  float maxError = 0.0f;
  for (int i =0 ; i < 4 ; i++){
    printf("y= %0.4f \n", y[i]);
  }

  for (int i = 0; i < N; i++)
    maxError = max(maxError, abs(y[i]-4.0f));
  printf("Max error: %f\n", maxError);

  hipFree(d_x);
  hipFree(d_y);
  free(x);
  free(y);
  return(1);
*/




